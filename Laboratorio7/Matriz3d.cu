#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#define BLOCK 10
using namespace std;

__global__
void matrizEscalar(float* A, float B, float* C, int fil,int col,int d3){
	int i = blockIdx.y*blockDim.y + threadIdx.y;
	int j = blockIdx.x*blockDim.x + threadIdx.x;
	int k = blockIdx.z*blockDim.z + threadIdx.z;
	if ((i < fil) && (j < col) && (k < d3)) {
		C[k*col*fil + i*col + j] = B * A[k*col*fil + i*col + j];
	}
}
void matrizescalar(float* A, float B, float* C, int fil,int col,int d3) {
	int size = fil * col * d3 *  sizeof(float);
	float *d_A, *d_C;
	hipMalloc((void **)&d_A, size);
	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
	hipMalloc((void **)&d_C, size);
	dim3 DimGrid((col-1)/BLOCK+1, (fil-1)/BLOCK+1,(d3-1)/BLOCK+1);
	dim3 DimBlock(BLOCK, BLOCK, BLOCK);
	matrizEscalar <<< dime1, DimBlock >>> (d_A, B, d_C, fil, col,d3);
	hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
	hipFree(d_A); hipFree(d_C);
}

int main() {
	int fila,columna,D3;
	float B;
	float *A;float *C;
	fila = 10;
	columna = 10;
	D3 = 10;
	B=2;	
	A = (float*)malloc(fila*columna*D3*sizeof(float));
	C = (float*)malloc(fila*columna*D3*sizeof(float));
	for (int i = 0; i < fila*columna*D3; i++)
		A[i] = 2;
	matrizescalar(A, B, C, fila,columna,D3);

}
