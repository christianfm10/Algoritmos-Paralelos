#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>




using namespace std;
__global__
void matrixAddKernel(float* A, float* B, float* C, int n){
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if (i < n*n) C[i] = A[i] + B[i];
}
__global__
void matrixAddKernel2(float* A, float* B, float* C, int n){
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if (i < n){
		for (int j = i * n; j < i * n + n; j++)
			C[j] = A[j] + B[j];
	}
}
__global__
void matrixAddKernel3(float* A, float* B, float* C, int n){
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if (i < n)
		for (int j = i; j < n*n; j += n)
			C[j] = A[j] + B[j];
}
void matrixAdd(float* A, float* B, float* C, int n) {
	int size = n * n * sizeof(float);
	float *d_A, *d_B, *d_C;
	hipMalloc((void **)&d_A, size);
	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
	hipMalloc((void **)&d_B, size);
	hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
	hipMalloc((void **)&d_C, size);
	matrixAddKernel <<< ceil((n*n) / 256.0), 256 >>> (d_A, d_B, d_C, n);
	hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
	hipFree(d_A); hipFree(d_B); hipFree(d_C);
}

int main() {
	int n;
	float *B;
	float *A;float *C;
	n=100;
	A = (float*)malloc(n*n*sizeof(float));
	B = (float*)malloc(n*sizeof(float));
	C = (float*)malloc(n*sizeof(float));
	for (int i = 0; i < n*n; i++)
	{
		A[i] = 1;
		B[i] = 2;
	}
	matrixAdd(A, B, C, n);

}
