#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

using namespace std;
__global__
void vecMultKernel(float* A, float* B, float* C, int n){
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if (i<n){
		C[i] = 0;
		for (int j = 0; j<n; j++)
			C[i] += A[i*n + j] * B[j];
	}
}
void matrizescalar(float* A, float* B, float* C, int n) {
	int size = n * n * sizeof(float);
	int sizevect = n * sizeof(float);
	float *d_A, *d_B, *d_C;
	hipMalloc((void **)&d_A, size);
	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
	hipMalloc((void **)&d_B, sizevect);
	hipMemcpy(d_B, B, sizevect, hipMemcpyHostToDevice);
	hipMalloc((void **)&d_C, sizevect);
	vecMultKernel <<< ceil((n*n) / 256.0), 256 >>> (d_A, d_B, d_C, n);
	hipMemcpy(C, d_C, sizevect, hipMemcpyDeviceToHost);
	hipFree(d_A); hipFree(d_B); hipFree(d_C);
}

int main() {
	int n;
	float *B;
	float *A;float *C;
	n=100;
	A = (float*)malloc(fila*columna*sizeof(float));
	B = (float*)malloc(fila*sizeof(float));
	C = (float*)malloc(fila*sizeof(float));
	for (int i = 0; i < fila*columna; i++)
		A[i] = 2;
	for (int i = 0; i < fila; i++)
		B[i] = 3;
	matrizescalar(A, B, C, n);

	
}
