#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#define BLOCK 8
using namespace std;

__global__
void matrizEscalar(float* A, float B, float* C, int fil,int col){
	int i = blockIdx.y*blockDim.y + threadIdx.y;
	int j = blockIdx.x*blockDim.x + threadIdx.x;
	if ((i < fil) && (j < col))
		C[i*col + j] = B * A[i*col + j];
	
}
void matrizescalar(float* A, float B, float* C, int fil,int col) {
	int size = fil * col * sizeof(float);
	float *d_A, *d_C;
	hipMalloc((void **)&d_A, size);
	hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
	hipMalloc((void **)&d_C, size);
	dim3 DimGrid((col-1)/BLOCK+1, (fil-1)/BLOCK+1);
	dim3 DimBlock(BLOCK, BLOCK);
	matrizEscalar <<< DimGrid, DimBlock >>> (d_A, B, d_C, fil, ccol);
	hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);
	hipFree(d_A); hipFree(d_C);
}
int main() {
	int fila,columna;
	float B;
	float *A;float *C;
	fila = 512;
	columna = 512;
	B = 4;
	A = (float*)malloc(fila*columna*sizeof(float));
	C = (float*)malloc(fila*columna*sizeof(float));
	for (int i = 0; i < fila*columna; i++)
		A[i] = 2;
	mostrarM(A, fila, columna);
	for (int i = 0; i < fila; i++){
		for (int j = 0; j < columna; j++)
			printf("%3.0f ", A[x*columna + y]);
		printf("\n");
	}
	matrizescalar(A, B, C, fila,columna);
	for (int i = 0; i < fila; i++){
		for (int j = 0; j < columna; j++)
			printf("%3.0f ", C[i*columna + j]);
		printf("\n");
	}
}

